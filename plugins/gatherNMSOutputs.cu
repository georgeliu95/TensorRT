#include "hip/hip_runtime.h"
#include <vector>

#include "ssd.h"
#include "ssdMacros.h"
#include "ssd_internal.h"

namespace nvinfer1
{
namespace plugin
{

template <typename T_BBOX, typename T_SCORE, unsigned nthds_per_cta>
__launch_bounds__(nthds_per_cta)
    __global__ void gatherNMSOutputs_kernel(
        const bool shareLocation,
        const int numImages,
        const int numPredsPerClass,
        const int numClasses,
        const int topK,
        const int keepTopK,
        const int* indices,
        const T_SCORE* scores,
        const T_BBOX* bboxData,
        int* numDetections,
        T_BBOX* nmsedBoxes,
        T_BBOX* nmsedScores,
        T_BBOX* nmsedClasses,
        bool clipBoxes
        )
{
    if (keepTopK > topK)
        return;
    for (int i = blockIdx.x * nthds_per_cta + threadIdx.x;
         i < numImages * keepTopK;
         i += gridDim.x * nthds_per_cta)
    {
        const int imgId = i / keepTopK;
        const int detId = i % keepTopK;
        const int offset = imgId * numClasses * topK;
        const int index = indices[offset + detId];
        const T_SCORE score = scores[offset + detId];
        if (index == -1)
        {
            nmsedClasses[i] = -1;
            nmsedScores[i] = 0;
            nmsedBoxes[i * 4] = 0;
            nmsedBoxes[i * 4 + 1] = 0;
            nmsedBoxes[i * 4 + 2] = 0;
            nmsedBoxes[i * 4 + 3] = 0;
        }
        else
        {
            const int bboxOffset = imgId * (shareLocation ? numPredsPerClass : (numClasses * numPredsPerClass));
            const int bboxId = ((shareLocation ? (index % numPredsPerClass)
                        : index % (numClasses * numPredsPerClass)) + bboxOffset) * 4;
            nmsedClasses[i] = (index % (numClasses * numPredsPerClass)) / numPredsPerClass; // label
            nmsedScores[i] = score;                                                        // confidence score
            // clipped bbox xmin
            nmsedBoxes[i * 4] = clipBoxes ? max(min(bboxData[bboxId],
                        T_BBOX(1.)), T_BBOX(0.)) : bboxData[bboxId];
            // clipped bbox ymin
            nmsedBoxes[i * 4 + 1] = clipBoxes ? max(min(bboxData[bboxId + 1],
                        T_BBOX(1.)), T_BBOX(0.)) : bboxData[bboxId + 1];
            // clipped bbox xmax
            nmsedBoxes[i * 4 + 2] = clipBoxes ? max(min(bboxData[bboxId + 2],
                        T_BBOX(1.)), T_BBOX(0.)) : bboxData[bboxId + 2];
            // clipped bbox ymax
            nmsedBoxes[i * 4 + 3] = clipBoxes ? max(min(bboxData[bboxId + 3],
                        T_BBOX(1.)), T_BBOX(0.)) : bboxData[bboxId + 3];
            atomicAdd(&numDetections[i / keepTopK], 1);
        }
    }
}

template <typename T_BBOX, typename T_SCORE>
ssdStatus_t gatherNMSOutputs_gpu(
    hipStream_t stream,
    const bool shareLocation,
    const int numImages,
    const int numPredsPerClass,
    const int numClasses,
    const int topK,
    const int keepTopK,
    const void* indices,
    const void* scores,
    const void* bboxData,
    void* numDetections,
    void* nmsedBoxes,
    void* nmsedScores,
    void* nmsedClasses,
    bool clipBoxes
    )
{
    hipMemsetAsync(numDetections, 0, numImages * sizeof(int), stream);
    const int BS = 32;
    const int GS = 32;
    gatherNMSOutputs_kernel<T_BBOX, T_SCORE, BS><<<GS, BS, 0, stream>>>(shareLocation, numImages, numPredsPerClass,
                                                                           numClasses, topK, keepTopK,
                                                                           (int*) indices, (T_SCORE*) scores, (T_BBOX*) bboxData,
                                                                           (int*) numDetections,
                                                                           (T_BBOX*) nmsedBoxes, 
                                                                           (T_BBOX*) nmsedScores, 
                                                                           (T_BBOX*) nmsedClasses,
                                                                           clipBoxes
                                                                            );

    CSC(hipGetLastError(), STATUS_FAILURE);
    return STATUS_SUCCESS;
}

// gatherNMSOutputs LAUNCH CONFIG {{{
typedef ssdStatus_t (*nmsOutFunc)(hipStream_t,
                               const bool,
                               const int,
                               const int,
                               const int,
                               const int,
                               const int,
                               const void*,
                               const void*,
                               const void*,
                               void*,
                               void*,
                               void*, 
                               void*,
                               bool);
struct nmsOutLaunchConfig
{
    DType_t t_bbox;
    DType_t t_score;
    nmsOutFunc function;

    nmsOutLaunchConfig(DType_t t_bbox, DType_t t_score)
        : t_bbox(t_bbox)
        , t_score(t_score)
    {
    }
    nmsOutLaunchConfig(DType_t t_bbox, DType_t t_score, nmsOutFunc function)
        : t_bbox(t_bbox)
        , t_score(t_score)
        , function(function)
    {
    }
    bool operator==(const nmsOutLaunchConfig& other)
    {
        return t_bbox == other.t_bbox && t_score == other.t_score;
    }
};

using nvinfer1::DataType;

static std::vector<nmsOutLaunchConfig> nmsOutFuncVec;

bool nmsOutputInit()
{
    nmsOutFuncVec.push_back(nmsOutLaunchConfig(DataType::kFLOAT, DataType::kFLOAT,
                                         gatherNMSOutputs_gpu<float, float>));
    return true;
}

static bool initialized = nmsOutputInit();

//}}}

ssdStatus_t gatherNMSOutputs(
    hipStream_t stream,
    const bool shareLocation,
    const int numImages,
    const int numPredsPerClass,
    const int numClasses,
    const int topK,
    const int keepTopK,
    const DType_t DT_BBOX,
    const DType_t DT_SCORE,
    const void* indices,
    const void* scores,
    const void* bboxData,
    void* numDetections,
    void* nmsedBoxes,
    void* nmsedScores,
    void* nmsedClasses,
    bool clipBoxes
    )
{
    nmsOutLaunchConfig lc = nmsOutLaunchConfig(DT_BBOX, DT_SCORE);
    for (unsigned i = 0; i < nmsOutFuncVec.size(); ++i)
    {
        if (lc == nmsOutFuncVec[i])
        {
            DEBUG_PRINTF("gatherNMSOutputs kernel %d\n", i);
            return nmsOutFuncVec[i].function(stream,
                                          shareLocation,
                                          numImages,
                                          numPredsPerClass,
                                          numClasses,
                                          topK,
                                          keepTopK,
                                          indices,
                                          scores,
                                          bboxData,
                                          numDetections,
                                          nmsedBoxes,
                                          nmsedScores,
                                          nmsedClasses,
                                          clipBoxes
                                          );
        }
    }
    return STATUS_BAD_PARAM;
}

} // namespace plugin
} // namespace nvinfer1
