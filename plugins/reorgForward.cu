#include "hip/hip_runtime.h"
#include "reducedMathPlugin.h"
#include "yolo.h"

using nvinfer1::plugin::reduced_divisor;

template <unsigned nthdsPerCTA>
__launch_bounds__(nthdsPerCTA)
    __global__ void reorgKernel(
        const float* input, // input tensor of shape (batch, C, H, W)
        const int volume,   // note that volumes of input and output tensors are the same
        reduced_divisor batch,
        reduced_divisor C,
        reduced_divisor H,
        reduced_divisor W,
        reduced_divisor C_out,
        reduced_divisor stride,
        float* output) // output tensor of shape (batch, C * stride * stride, H / stride, W / stride)
{

    // outIndex is row-major position of input coordinates
    for (int outIndex = blockIdx.x * nthdsPerCTA + threadIdx.x; outIndex < volume; outIndex += nthdsPerCTA)
    {
        int i = outIndex;

        // calculate output coordinates from outIndex
        int outW, outH, outC;
        W.divmod(i, i, outW);
        H.divmod(i, i, outH);
        C.divmod(i, i, outC);
        int outN = i;

        // calculate input coordinates based on output coordinates
        // offset is [0, 1, ..., stride * stride - 1] = posH * stride + posW
        int offset, inC, posH, posW;
        C_out.divmod(outC, offset, inC);
        stride.divmod(offset, posH, posW);
        int inH = outH * stride.get() + posH;
        int inW = outW * stride.get() + posW;
        int inN = outN;

        // inIndex is row-major position of input coordinates
        int inIndex = inW + W.get() * stride.get() * (inH + H.get() * stride.get() * (inC + C_out.get() * inN));

        output[outIndex] = input[inIndex];
    }
}

yoloStatus_t reorgGPU(
    hipStream_t stream,
    const int batch,
    const int C,
    const int H,
    const int W,
    const int stride,
    const float* input,
    float* output)
{
    const int BS = 512;                    // number of threads in one block
    const int volume = batch * C * H * W;  // size of input tensor
    const int GS = (volume + BS - 1) / BS; // number of blocks to launch, calculated so global number of threads is >= volume

    reduced_divisor C_out(C / (stride * stride));
    reorgKernel<BS><<<GS, BS, 0, stream>>>(input, volume, reduced_divisor(batch), reduced_divisor(C), reduced_divisor(H), reduced_divisor(W), C_out, reduced_divisor(stride), output);
    return STATUS_SUCCESS;
}

yoloStatus_t reorgInference(
    hipStream_t stream,
    const int batch,
    const int C,
    const int H,
    const int W,
    const int stride,
    const void* input,
    void* output)
{
    return reorgGPU(stream, batch, C, H, W, stride, (const float*) input, (float*) output);
}
