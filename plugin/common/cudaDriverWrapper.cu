#ifdef __linux__
#ifdef __x86_64__
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <dlfcn.h>
#include "cudaDriverWrapper.h"
#include "checkMacros.h"
#include "helpers.h"

using namespace nvinfer1;

CUDADriverWrapper::CUDADriverWrapper()
{
    handle = dlopen("libcuda.so.1", RTLD_LAZY);
    ASSERT(handle != nullptr);

    auto load_sym = [](void *handle, const char *name) {
        void *ret = dlsym(handle, name);
        ASSERT(ret != nullptr);
        return ret;
    };

    *(void**)(&_cuGetErrorName) = load_sym(handle, "hipDrvGetErrorName");
    *(void**)(&_cuFuncSetAttribute) = load_sym(handle, "cuFuncSetAttribute");
    *(void**)(&_cuLinkComplete) = load_sym(handle, "hiprtcLinkComplete");
    *(void**)(&_cuModuleUnload) = load_sym(handle, "hipModuleUnload");
    *(void**)(&_cuLinkDestroy) = load_sym(handle, "hiprtcLinkDestroy");
    *(void**)(&_cuModuleLoadData) = load_sym(handle, "hipModuleLoadData");
    *(void**)(&_cuLinkCreate) = load_sym(handle, "hiprtcLinkCreate");
    *(void**)(&_cuModuleGetFunction) = load_sym(handle, "hipModuleGetFunction");
    *(void**)(&_cuLinkAddFile) = load_sym(handle, "hiprtcLinkAddFile");
    *(void**)(&_cuLinkAddData) = load_sym(handle, "hiprtcLinkAddData");
    *(void**)(&_cuLaunchCooperativeKernel) = load_sym(handle, "hipModuleLaunchCooperativeKernel");
}

CUDADriverWrapper::~CUDADriverWrapper()
{
    dlclose(handle);
}

hipError_t CUDADriverWrapper::hipDrvGetErrorName(hipError_t error, const char** pStr) const
{
    return (*_cuGetErrorName)(error, pStr);
}

hipError_t CUDADriverWrapper::cuFuncSetAttribute(hipFunction_t hfunc, hipFunction_attribute attrib, int  value) const
{
    return (*_cuFuncSetAttribute)(hfunc, attrib, value);
}

hipError_t CUDADriverWrapper::hiprtcLinkComplete(hiprtcLinkState state, void** cubinOut, size_t* sizeOut) const
{
    return (*_cuLinkComplete)(state, cubinOut, sizeOut);
}

hipError_t CUDADriverWrapper::hipModuleUnload(hipModule_t hmod) const
{
    return (*_cuModuleUnload)(hmod);
}

hipError_t CUDADriverWrapper::hiprtcLinkDestroy(hiprtcLinkState state) const
{
    return (*_cuLinkDestroy)(state);
}

hipError_t CUDADriverWrapper::hipModuleLoadData(hipModule_t* module, const void* image) const
{
    return (*_cuModuleLoadData)(module, image);
}

hipError_t CUDADriverWrapper::hiprtcLinkCreate(unsigned int numOptions, hipJitOption* options, void** optionValues, hiprtcLinkState* stateOut) const
{
    return (*_cuLinkCreate)(numOptions, options, optionValues, stateOut);
}

hipError_t CUDADriverWrapper::hipModuleGetFunction(hipFunction_t* hfunc, hipModule_t hmod, const char* name) const
{
    return (*_cuModuleGetFunction)(hfunc, hmod, name);
}

hipError_t CUDADriverWrapper::hiprtcLinkAddFile(hiprtcLinkState state, hipJitInputType type, const char* path, unsigned int numOptions, hipJitOption* options, void** optionValues) const
{
    return (*_cuLinkAddFile)(state, type, path, numOptions, options, optionValues);
}

hipError_t CUDADriverWrapper::hiprtcLinkAddData(hiprtcLinkState state, hipJitInputType type, void* data, size_t size, const char* name, unsigned int numOptions, hipJitOption* options, void** optionValues) const
{
    return (*_cuLinkAddData)(state, type, data, size, name, numOptions, options, optionValues);
}

hipError_t CUDADriverWrapper::hipModuleLaunchCooperativeKernel (hipFunction_t f, unsigned int  gridDimX, unsigned int  gridDimY, unsigned int  gridDimZ,
    unsigned int  blockDimX, unsigned int  blockDimY, unsigned int  blockDimZ, unsigned int  sharedMemBytes, hipStream_t hStream, void** kernelParams) const
{
    return (*_cuLaunchCooperativeKernel)(f, gridDimX, gridDimY, gridDimZ,
        blockDimX, blockDimY, blockDimZ, sharedMemBytes, hStream, kernelParams);
}

#endif // __x86_64__
#endif //__linux__
